#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include "utils.h"
#include "device_utils.h"
#include "main_utils.cu"
#ifndef MAIN_H
#include "main.h"
#endif

#define PRINT_SUMMARY 1
#define PRINT_MAIN_LOOP 0
#define PRINT_WORST 1



int main(){

	//allocate graph
	float **g = graph_generate(N_NODES);
	print_graph(g, N_NODES);
	float **m= graph_to_mat(g, N_NODES);
	float *vec_graph = mat_to_vec(m, N_NODES);
	float *d_vec_graph;
	hipMalloc((void **) &d_vec_graph, N_NODES*N_NODES*sizeof(float));
	hipMemcpy( d_vec_graph,  vec_graph, N_NODES*N_NODES*sizeof(float), hipMemcpyHostToDevice);

	free(g);
	free(m);

	//allocate data arrays			DIM
	int * d_population; 			//POPULATION_SIZE*N_NODES
	
	
	int *d_offspring;			//POPULATION_SIZE*N_NODES*OFFSPRING_FACTOR

	float *d_fitness;			//POPULATION_SIZE*OFFSPRING_FACTOR
	int *d_auxiliary;			

	unsigned int *d_shuffle_rands;		//POPULATION_SIZE
	
	unsigned int *d_genetic_rands;		//N_WARPS*OFFSPRING_FACTOR*3
	

	CUDA_CALL(hipMalloc((void **) &d_population, POPULATION_SIZE*N_NODES*sizeof(int)));
	CUDA_CALL(hipMalloc((void **) &d_offspring, POPULATION_SIZE*N_NODES*OFFSPRING_FACTOR*sizeof(int)));
	CUDA_CALL(hipMalloc((void **) &d_auxiliary, POPULATION_SIZE*OFFSPRING_FACTOR*sizeof(int)));
	CUDA_CALL(hipMalloc((void **) &d_fitness, POPULATION_SIZE*OFFSPRING_FACTOR*sizeof(float)));
	CUDA_CALL(hipMalloc((void **) &d_shuffle_rands, POPULATION_SIZE*sizeof(unsigned int)));

	int n_warps = POPULATION_SIZE/32;
	if(POPULATION_SIZE <32){
		n_warps =1;
	}

	CUDA_CALL(hipMalloc((void **) &d_genetic_rands, n_warps*OFFSPRING_FACTOR*3*sizeof(unsigned int)));
		
	
	
	//create hiprand generator
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, 1234);

	
	//initialize the data
	CUDA_CALL(init_population(d_population,gen, N_NODES, POPULATION_SIZE));

#if DEBUG
	int *pop = (int*) malloc(N_NODES*POPULATION_SIZE*sizeof(int));
	CUDA_CALL(hipMemcpy( pop, d_population, N_NODES*POPULATION_SIZE*sizeof(int), hipMemcpyDeviceToHost));

	for(int t=0; t<POPULATION_SIZE; ++t){
		for(int s =0; s<N_NODES; ++s){
			printf("%d ", pop[t*N_NODES + s]);
		}
		printf("\n");
	}
	int *off = (int*) malloc(N_NODES*POPULATION_SIZE*OFFSPRING_FACTOR*sizeof(int));
#endif
	
	//kernel parameters
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);

	dim3 threadsP(32,THREADS_PER_BLOCK/32,1);
	dim3 blocksP(ceil(POPULATION_SIZE/THREADS_PER_BLOCK),1,1); 
	if(POPULATION_SIZE< THREADS_PER_BLOCK){ 
		threadsP.y = (POPULATION_SIZE < 32 ? 1 : POPULATION_SIZE/32);
		blocksP.x = 1;
	}
	if(POPULATION_SIZE<32 ){
		threadsP.x = POPULATION_SIZE;
	}


	dim3 threadsS(32,THREADS_PER_BLOCK/32,1);
	dim3 blocksS(ceil((POPULATION_SIZE*OFFSPRING_FACTOR)/THREADS_PER_BLOCK),1,1); 
	if(POPULATION_SIZE*OFFSPRING_FACTOR< THREADS_PER_BLOCK){ 
		threadsS.y = (POPULATION_SIZE*OFFSPRING_FACTOR < 32 ? 1 : POPULATION_SIZE*OFFSPRING_FACTOR/32);
		blocksS.x = 1;
	}
	if(POPULATION_SIZE*OFFSPRING_FACTOR<32 ){
		threadsS.x = POPULATION_SIZE*OFFSPRING_FACTOR;
	}
	if(THREADS_PER_BLOCK < 32){
		threadsS.x = THREADS_PER_BLOCK;
		threadsS.y = 1;
		threadsP.x = THREADS_PER_BLOCK;
		threadsP.y = 1;
	}

	printf("operation on population will be launched on %d blocks with dim (%d, %d)\n", blocksP.x, threadsP.x,threadsP.y);
	printf("operation on offspring will be launched on %d blocks with dim (%d, %d)\n", blocksS.x, threadsS.x,threadsS.y);

	//support variables
	int *global_best_sol = (int*) malloc(N_NODES*sizeof(int));
	float best_fitness = FLT_MAX;
	float current_fitness;	
	float fitnesses[N_ITERATIONS];

	int *d_global_best_sol;
	float *d_best_fitness;
	CUDA_CALL(hipMalloc((void **) &d_global_best_sol, N_NODES*sizeof(int)));
	CUDA_CALL(hipMalloc((void **) &d_best_fitness, sizeof(float)));
	hipMemcpy( d_best_fitness, &best_fitness, sizeof(float), hipMemcpyHostToDevice);
	
	//use events for measuring performance
	hipEvent_t start, stop;

	CUDA_CALL(hipEventCreate(&start));
	CUDA_CALL(hipEventCreate(&stop));
	CUDA_CALL(hipEventRecord(start, 0));
	
	//main loop
	for(int t=0; t<N_ITERATIONS; ++t){
		//generate random numbers for offspring generation
		hiprandGenerate(gen, (unsigned int *) d_genetic_rands, n_warps*OFFSPRING_FACTOR*3*sizeof(unsigned int));
#if PRINT_MAIN_LOOP
		printf("it %d: generating the offspring\n", t);
#endif
		naive_generation<<<blocksP, threadsP>>>(d_population, 
							d_offspring, 
							d_genetic_rands);
	
#if PRINT_MAIN_LOOP		
		printf("it %d: applying selection\n", t);
#endif
		naive_selection(d_offspring,
				d_population,
				N_NODES,
				POPULATION_SIZE,
				OFFSPRING_FACTOR,
				d_vec_graph,
				d_fitness,
				d_auxiliary,
				threadsS,
				blocksS,
				threadsP,
				blocksP);


		//swap if better than global best
		swap_best<<<1, N_NODES>>>(	d_population, 
						d_fitness, 
						0, 
						d_global_best_sol, 
						d_best_fitness);

		hipMemcpy( &current_fitness, d_fitness, sizeof(float), hipMemcpyDeviceToHost);
		fitnesses[t] = current_fitness;
#if PRINT_MAIN_LOOP		
		printf("it %d: currently found fitness is %.2f\n", t, current_fitness);
#endif

		//shuffle
		thrust_shuffle(d_population, d_offspring, d_auxiliary, gen, d_shuffle_rands, N_NODES, POPULATION_SIZE);
				

	}
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("Iterated %d times, elapsed time is %.2f ms, for %.2f ms/it\n", N_ITERATIONS, elapsedTime, elapsedTime/N_ITERATIONS);

#if PRINT_SUMMARY
	printf("summary of iterations:\n");

	for(int t=0; t<N_ITERATIONS; ++t){
		printf("%.2f ->", fitnesses[t]);
	}
	printf("\n");
#endif

#if PRINT_WORST
	printf("printing the worst solution as a metric for diversity in the population:\n");
	int *worst = (int*) malloc(N_NODES*sizeof(int));
	hipMemcpy( worst, d_population+N_NODES*(POPULATION_SIZE-1), N_NODES*sizeof(int), hipMemcpyDeviceToHost);
	for(int t=0; t<N_NODES; ++t){
		printf("%d ", worst[t]);
		if(t%10 ==0) printf("\n");
	}
	printf("\n");
#endif
	
	hipMemcpy( global_best_sol, d_global_best_sol, N_NODES*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy( &best_fitness, d_best_fitness, sizeof(float), hipMemcpyDeviceToHost);
	printf("best solution found has path length %.2f\n", best_fitness);

	for(int t=0; t<N_NODES; ++t){
		printf("%d ->", global_best_sol[t]);
	}
	printf("\n");


	hipEventDestroy(start);
	hipEventDestroy(stop);
		
	free(vec_graph);
	free(global_best_sol);
	
	hipFree(d_population);
	hipFree(d_offspring);
	hipFree(d_shuffle_rands);
	hipFree(d_genetic_rands);
	hipFree(d_auxiliary);
	hipFree(d_global_best_sol);
	hipFree(d_best_fitness);

	hiprandDestroyGenerator(gen);
#if DEBUG
	free(pop);
	free(off);
#endif

	hipDeviceReset();

}















