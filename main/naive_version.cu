#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include "utils.h"
#include "device_utils.h"
#include "kernels.h"
#include "main.h"

int * init_population(hiprandGenerator_t gen, int n_dim, int population_dim){
	int *pop;
	unsigned int * rands;

	
	hipMalloc((void **) &pop, n_dim*population_dim*sizeof(unsigned int));
	hipMalloc((void **) &rands, n_dim*population_dim*n_dim*sizeof(unsigned int));
	
	hiprandGenerate(gen, (unsigned int *) rands, population_dim*n_dim*sizeof(unsigned int));
	
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
	
	dim3 threads(32,prop.maxThreadsDim[0]/32,1);
	dim3 blocks(ceil(population_dim/prop.maxThreadsDim[0]),1,1); 
	if(population_dim< prop.maxThreadsDim[0]){ 
		threads.y = (population_dim < 32 ? 1 : population_dim/32);
		blocks.x = 1;
	}
	if(population_dim<32 ){
		threads.x = population_dim;
	}

	init_pop_s<<<blocks, threads>>>(pop, population_dim, n_dim, rands);
	
	hipFree(rands);
	return pop;

}

