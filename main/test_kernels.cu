#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include "utils.h"
#include "device_utils.h"
#include "kernels.h"
#include "main.h"

void test_init();
void test_ngen();

int main(void){
	test_init();
	return 0;
}

void test_init(){
	int *pop;
	int *d_pop;
	unsigned int *d_rands;
	
	pop= (int*) malloc(POPULATION_SIZE*N_NODES*sizeof(int));
	hipMalloc( (void **) &d_pop, POPULATION_SIZE*N_NODES*sizeof(int) );

	hipMalloc((void **) &d_rands, POPULATION_SIZE*N_NODES*sizeof(unsigned int));
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, 1234);
	hiprandGenerate(gen, (unsigned int *) d_rands, POPULATION_SIZE*N_NODES*sizeof(unsigned int));

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
	
	dim3 threads(32,prop.maxThreadsDim[0]/32,1);
	dim3 blocks(ceil(POPULATION_SIZE/prop.maxThreadsDim[0]),1,1); 
	if(POPULATION_SIZE< prop.maxThreadsDim[0]){ 
		threads.y = POPULATION_SIZE/32;
		blocks.x = 1;
	}	
	

	printf("launching with (%d, %d,%d) threads and %d blocks\n", 
			threads.x, threads.y, threads.z, blocks.x);
	init_pop_s<<<blocks, threads>>>(d_pop, POPULATION_SIZE, N_NODES, d_rands, (POPULATION_SIZE/32)*3);
	
	hipMemcpy( pop, d_pop, POPULATION_SIZE*N_NODES*sizeof(int), hipMemcpyDeviceToHost);
	
	for(int t =0; t<POPULATION_SIZE; ++t){
		for(int s =0; s<N_NODES; ++s){
			printf("%d ", pop[t*N_NODES + s]);
		}
		printf("\n");
	}
	free(pop);
	hiprandDestroyGenerator(gen);
	hipFree(d_rands);
	hipFree(d_pop);
}

void test_ngen(){
	/*
	int *pop;
	int *d_pop, d_off;

	unsigned int *rands;
	
	pop= (int*) malloc(POPULATION_SIZE*N_NODES*sizeof(int));	
	*/
	
}

