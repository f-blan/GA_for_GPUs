#include <stdio.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include "utils.h"
#include "device_utils.h"
#include "kernels.h"
#include "main.h"



void test_utils();
void test_gen_ops();
void test_curand();

int main(void){
	//test_utils();
	//test_gen_ops();
	test_curand();

	return 0;
}

void test_utils(){
	float **g = graph_generate(N_NODES);
	print_graph(g, N_NODES);
	
	float **m = graph_to_mat(g, N_NODES);
	//print_mat(m, N_NODES);
}

void test_gen_ops(){
	int vec[] = {0,1,2,3,4};
	swap_mutation(vec);

}

void test_curand(){
	int size = 32;

	hiprandGenerator_t gen;

	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

	int *d_r; 
	hipMalloc((void **) &d_r, size*sizeof(int));

	int *r =(int *) malloc(size*sizeof(int));

	hiprandSetPseudoRandomGeneratorSeed(gen, 1234);

	hiprandGenerate(gen, (unsigned int *) d_r, size*sizeof(int));
	
	hipMemcpy(r, d_r, size*sizeof(int), hipMemcpyDeviceToHost);
	
	for(int i =0; i<size; ++i){
		printf("%d ", r[i]);
	}
	printf("\n");
	
	hiprandDestroyGenerator(gen);
	hipFree(d_r);
	free(r);
	
}











