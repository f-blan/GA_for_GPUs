#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include "kernels.h"

void print_pop(int *d_pop, int pop_dim){
	int *pop = (int *) malloc(pop_dim *N_NODES* sizeof(int));
	hipMemcpy( pop, d_pop, pop_dim*N_NODES*sizeof(int), hipMemcpyDeviceToHost);
	for(int s=0; s< pop_dim; ++s){
		for(int k =0 ; k<N_NODES; ++k){			
			printf("%d ", pop[s*N_NODES + k]);
		}
		printf("\n");
	}

	free(pop);
}

void print_popfit(int *d_pop,float * d_fit, int pop_dim){
	int *pop = (int *) malloc(pop_dim *N_NODES* sizeof(int));
	float * fit = (float *) malloc(pop_dim * sizeof(float));
	hipMemcpy( pop, d_pop, pop_dim*N_NODES*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy( fit, d_fit, POPULATION_SIZE*sizeof(float), hipMemcpyDeviceToHost);
	for(int s=0; s< pop_dim; ++s){
		printf("%d) ", s);
		for(int k =0 ; k<N_NODES; ++k){			
			printf("%d ", pop[s*N_NODES + k]);
		}
		printf("- %.2f\n", fit[s]);
	}

	free(pop);
}

int init_population(int * pop,hiprandGenerator_t gen, int n_dim, int population_dim){
	
	unsigned int * rands;

	
	hipMalloc((void **) &rands, population_dim*n_dim*sizeof(unsigned int));
	
	hiprandGenerate(gen, (unsigned int *) rands, population_dim*n_dim*sizeof(unsigned int));
	
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
	
 
        dim3 threads(32,prop.maxThreadsDim[0]/32,1);
	dim3 blocks(ceil(population_dim/prop.maxThreadsDim[0]),1,1); 
	if(population_dim< prop.maxThreadsDim[0]){ 
		threads.y = population_dim/32;
		blocks.x = 1;
	}
        if(population_dim<32 ){
                threads.x = population_dim;
		threads.y = 1;
        }

	
	
	init_pop_s<<<blocks, threads>>>(pop, population_dim, n_dim, rands);
	hipDeviceSynchronize();
	
	
	hipFree(rands);
	return hipSuccess;

}

