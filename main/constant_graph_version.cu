#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include "utils.h"
#include "device_utils.h"
#include "main_utils.cu"
#ifndef MAIN_H
#include "main.h"
#endif

#define PRINT_SUMMARY 1
#define DEBUG 0
#define DEBUG_BLOCK 0
#define PROVIDE_SOL 0
#define PRINT_MAIN_LOOP 0
#define PRINT_WORST 1


int main(void){
	float **g = graph_generate(N_NODES);
	print_graph(g, N_NODES);
	float **m= graph_to_mat(g, N_NODES);
	float *vec_graph = mat_to_vec(m, N_NODES);
	

	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(const_graph), vec_graph, N_NODES*N_NODES*sizeof(float)));

	free(g);
	free(m);
	
	//support variables
	int *global_best_sol = (int*) malloc(N_NODES*sizeof(int));
	float best_fitness = FLT_MAX;
	float current_fitness;	
	float fitnesses[N_ITERATIONS];

	int *d_global_best_sol;
	float *d_best_fitness;
	CUDA_CALL(hipMalloc((void **) &d_global_best_sol, N_NODES*sizeof(int)));
	CUDA_CALL(hipMalloc((void **) &d_best_fitness, sizeof(float)));
	hipMemcpy( d_best_fitness, &best_fitness, sizeof(float), hipMemcpyHostToDevice);

	//allocate data arrays			DIM
	int * d_population; 			//POPULATION_SIZE*N_NODES
	
	
	int *d_offspring;			//POPULATION_SIZE*N_NODES*OFFSPRING_FACTOR

	float *d_fitness;			//POPULATION_SIZE*OFFSPRING_FACTOR
	int *d_auxiliary;			

	unsigned int *d_shuffle_rands;		//POPULATION_SIZE
	
	unsigned int *d_genetic_rands;		//N_WARPS*OFFSPRING_FACTOR*3

	CUDA_CALL(hipMalloc((void **) &d_population, POPULATION_SIZE*N_NODES*sizeof(int)));
	CUDA_CALL(hipMalloc((void **) &d_offspring, POPULATION_SIZE*N_NODES*OFFSPRING_FACTOR*sizeof(int)));
	CUDA_CALL(hipMalloc((void **) &d_auxiliary, POPULATION_SIZE*OFFSPRING_FACTOR*sizeof(int)));
	CUDA_CALL(hipMalloc((void **) &d_fitness, POPULATION_SIZE*OFFSPRING_FACTOR*sizeof(float)));
	CUDA_CALL(hipMalloc((void **) &d_shuffle_rands, POPULATION_SIZE*sizeof(unsigned int)));

	int n_warps = POPULATION_SIZE/32;
	if(POPULATION_SIZE <32){
		n_warps =1;
	}

	CUDA_CALL(hipMalloc((void **) &d_genetic_rands, n_warps*OFFSPRING_FACTOR*3*sizeof(unsigned int)));

	//create hiprand generator
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, 1234);

	
	//initialize the data
	printf("initializing population\n");
	CUDA_CALL(init_population(d_population,gen, N_NODES, POPULATION_SIZE));

	//kernel parameters
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);

	dim3 threadsP(32,THREADS_PER_BLOCK/32,1);
	dim3 blocksP(ceil(POPULATION_SIZE/THREADS_PER_BLOCK),1,1); 
	if(POPULATION_SIZE< THREADS_PER_BLOCK){ 
		threadsP.y = (POPULATION_SIZE < 32 ? 1 : POPULATION_SIZE/32);
		blocksP.x = 1;
	}
	if(POPULATION_SIZE<32 ){
		threadsP.x = POPULATION_SIZE;
	}

	dim3 threadsS(32,THREADS_PER_BLOCK/32,1);
	dim3 blocksS(ceil((POPULATION_SIZE*OFFSPRING_FACTOR)/THREADS_PER_BLOCK),1,1); 
	if(POPULATION_SIZE*OFFSPRING_FACTOR< THREADS_PER_BLOCK){ 
		threadsS.y = (POPULATION_SIZE*OFFSPRING_FACTOR < 32 ? 1 : POPULATION_SIZE*OFFSPRING_FACTOR/32);
		blocksS.x = 1;
	}
	if(POPULATION_SIZE*OFFSPRING_FACTOR<32 ){
		threadsS.x = POPULATION_SIZE*OFFSPRING_FACTOR;
	}

	if(THREADS_PER_BLOCK < 32){
		threadsS.x = THREADS_PER_BLOCK;
		threadsS.y = 1;
		threadsP.x = THREADS_PER_BLOCK;
		threadsP.y = 1;
	}

	printf("operation on population will be launched on %d blocks with dim (%d, %d)\n", blocksP.x, threadsP.x,threadsP.y);
	printf("operation on offspring will be launched on %d blocks with dim (%d, %d)\n", blocksS.x, threadsS.x,threadsS.y);


	//use events for measuring performance
	hipEvent_t start, stop;

	CUDA_CALL(hipEventCreate(&start));
	CUDA_CALL(hipEventCreate(&stop));
	CUDA_CALL(hipEventRecord(start, 0));

	//main loop
	printf("running main loop\n");
	for(int t=0; t<N_ITERATIONS; ++t){
		//generate random numbers for offspring generation
		hiprandGenerate(gen, (unsigned int *) d_genetic_rands, n_warps*OFFSPRING_FACTOR*3*sizeof(unsigned int));
#if PRINT_MAIN_LOOP
		printf("it %d: generating the offspring\n", t);
#endif
		naive_generation<<<blocksP, threadsP>>>(d_population, 
							d_offspring, 
							d_genetic_rands);
	
#if PRINT_MAIN_LOOP		
		printf("it %d: applying selection\n", t);
#endif
		const_selection(d_offspring,
				d_population,
				N_NODES,
				POPULATION_SIZE,
				OFFSPRING_FACTOR,
				d_fitness,
				d_auxiliary,
				threadsS,
				blocksS,
				threadsP,
				blocksP);


		//swap if better than global best
		swap_best<<<1, N_NODES>>>(	d_population, 
						d_fitness, 
						0, 
						d_global_best_sol, 
						d_best_fitness);

		hipMemcpy( &current_fitness, d_fitness, sizeof(float), hipMemcpyDeviceToHost);
		fitnesses[t] = current_fitness;
#if PRINT_MAIN_LOOP		
		printf("it %d: currently found fitness is %.2f\n", t, current_fitness);
#endif
		fitnesses[t] = current_fitness;


		//shuffle
		thrust_shuffle(d_population, d_offspring, d_auxiliary, gen, d_shuffle_rands, N_NODES, POPULATION_SIZE);
				

	}


	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("Iterated %d times, elapsed time is %.2f ms, for %.2f ms/it\n", N_ITERATIONS, elapsedTime, elapsedTime/N_ITERATIONS);

#if PRINT_SUMMARY
	printf("summary of iterations:\n");

	for(int t=0; t<N_ITERATIONS; ++t){
		printf("%.2f ->", fitnesses[t]);
		if(t%10 ==0) printf("\n");
	}
	printf("\n");
#endif

#if PRINT_WORST
	printf("printing the worst solution as a metric for diversity in the population:\n");
	int *worst = (int*) malloc(N_NODES*sizeof(int));
	hipMemcpy( worst, d_population+N_NODES*(POPULATION_SIZE-1), N_NODES*sizeof(int), hipMemcpyDeviceToHost);
	for(int t=0; t<N_NODES; ++t){
		printf("%d ", worst[t]);
	}
	printf("\n");
#endif
	
	hipMemcpy( global_best_sol, d_global_best_sol, N_NODES*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy( &best_fitness, d_best_fitness, sizeof(float), hipMemcpyDeviceToHost);
	printf("best solution found has path length %.2f\n", best_fitness);

	for(int t=0; t<N_NODES; ++t){
		printf("%d ->", global_best_sol[t]);
	}
	printf("\n");

	hipEventDestroy(start);
	hipEventDestroy(stop);
		
	free(vec_graph);
	free(global_best_sol);
	hiprandDestroyGenerator(gen);
	
	hipFree(d_population);
	hipFree(d_offspring);
	hipFree(d_shuffle_rands);
	hipFree(d_genetic_rands);
	hipFree(d_auxiliary);
	hipFree(d_global_best_sol);
	hipFree(d_best_fitness);
#if DEBUG
	free(pop);
	free(off);
#endif

	hipDeviceReset();


}










