#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>

#define SWAP_ITERATIONS 2
#define DEBUG_PRINT 0




//simple function to initialize the population pseudo randomly

//this performs a bubble sort based on the random numbers generated, and produces the related permutation
//on the individual
__global__ void init_pop_s(int *pop, int pop_dim, int n_dim, unsigned int *random_nums){

	int tid = blockIdx.x*(blockDim.x*blockDim.y) + threadIdx.y*blockDim.x+ threadIdx.x;

	//pseudo random initialization based on tid + random_nums
	for(int t =0; t<n_dim; ++t)
		pop[tid*n_dim + t] = t;

	
	unsigned int random_start = tid*n_dim;
	int tmp;
	for(int t=0; t<n_dim-1; ++t){
		for(int s =0; s<n_dim-1-t; ++s){
			if(random_nums[random_start + s] > random_nums[random_start +s +1]){
				tmp = random_nums[random_start +s ];
				random_nums[random_start +s] = random_nums[random_start +s +1];
				random_nums[random_start +s +1] = tmp;
				
				tmp = pop[tid*n_dim +s ];
				pop[tid*n_dim +s] = pop[tid*n_dim +s +1];
				pop[tid*n_dim +s +1] = tmp;
			}
		}
	}
}

//simple shuffle algorithm

__global__ void shuffle(int *population,int*out, int population_dim, int n_dim, int*auxiliary, unsigned int *rands){
	unsigned int tid = blockIdx.x*(blockDim.x*blockDim.y) + threadIdx.y*blockDim.x+ threadIdx.x;	
	
	unsigned int tid_idx,tmp;
	unsigned int d_0, d_1;
	unsigned int offset = 0;
	unsigned int tid_max = (population_dim -1);
	int i;
	auxiliary[tid] = tid;



	__syncthreads();
	//sort the children
	for(i = 0; i< population_dim; ++i){
		tid_idx = (tid*2) + offset;
		if(tid_idx < tid_max){
			d_0 = rands[tid_idx];
			d_1 = rands[tid_idx + 1];
			if(d_0 < d_1){
				rands[tid_idx] = d_1;
				rands[tid_idx+1] = d_0;
				
				tmp = auxiliary[tid_idx];
				auxiliary[tid_idx] = auxiliary[tid_idx+1];
				auxiliary[tid_idx+1] = tmp;

			}
		}
		if(offset == 0){
			offset = 1;
		}else{
			offset = 0;
		}
		__syncthreads();
	}
	int val;
	for(i=0; i<n_dim; ++i){
		
		out[tid*n_dim + i]=population[n_dim*auxiliary[tid] + i];
	}
}
__global__ void swap_with_positions(int *copy, int *out, int *positions, int n_dim, int population_dim){
	unsigned int tid = blockIdx.x*(blockDim.x*blockDim.y) + threadIdx.y*blockDim.x+ threadIdx.x;

	int pos = positions[tid];

	for(int t=0; t<n_dim; ++t){
		out[tid*n_dim + t] = copy[pos*n_dim+t];
	}

}

//a way to randomly shuffle the population. Requires some auxiliary vectors
void thrust_shuffle(int *pop,int * copy, int *positions, hiprandGenerator_t gen, unsigned int * rands, int n_dim, int population_dim){
	
	hipMemcpy( copy, pop, population_dim*n_dim*sizeof(int), hipMemcpyDeviceToDevice);
	
	hiprandGenerate(gen, (unsigned int *) rands, population_dim*sizeof(unsigned int));


	thrust::sort_by_key(thrust::device, rands, rands+population_dim, positions);
	


	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,0);
	
	dim3 threads(32,prop.maxThreadsDim[0]/32,1);
	dim3 blocks(ceil(population_dim/prop.maxThreadsDim[0]),1,1); 
	if(population_dim< prop.maxThreadsDim[0]){ 
		threads.y = (population_dim < 32 ? 1 : population_dim/32);
		blocks.x = 1;
	}
	if(population_dim<32 ){
		threads.x = population_dim;
	}
	swap_with_positions<<<blocks, threads>>>(  copy, pop, positions, n_dim, population_dim);
}

__global__ void swap_best(int *population, float * fitness, int best_pos, int *global_best_sol, float *global_best_fit){
	
	int tid = threadIdx.x;
	if(fitness[best_pos] < global_best_fit[0]){
		global_best_sol[tid] = population[best_pos*N_NODES + tid];
		if(tid ==0){
			global_best_fit[0] = fitness[best_pos];
		}
	}


}




